#include "hip/hip_runtime.h"
#include "RabinKarpSearch.cuh"
#include "StandardHash.cuh"

#include <tuple>
#include "stdio.h"
#include <iostream>

namespace {
    constexpr auto PRIME = 23;
    constexpr auto BASE = 36; //input alphabet's length
}

RabinKarpSearch::RabinKarpSearch(std::string file, std::string pattern, std::unique_ptr<Hash> hash, std::unique_ptr<TextSplitter> textSplitter): file(std::move(file)), pattern(std::move(pattern)), hash(std::move(hash)), textSplitter(std::move(textSplitter))
{
    init();
    initGPU();
}

RabinKarpSearch::~RabinKarpSearch() {
    hipFree(umRanges);
    hipFree(umText);
    hipFree(umPattern);
}

void RabinKarpSearch::init()
{
    reader = TextReaderFactory::forText(file);
    if(!reader)
        throw std::runtime_error("Couldn't create a reader!");
    text = reader->read();
    calculateHashes();
    for(size_t i = 1; i < pattern.length(); ++i)
    {
        mostSignificantWeight *= BASE;
    }
}

void RabinKarpSearch::initGPU() {
    // Allocate unified memory for ranges
    std::vector<std::pair<size_t, size_t>> ranges = textSplitter->splitText(text.length(), pattern.length());
    hipMallocManaged(&umRanges, ranges.size() * sizeof(std::pair<int, int>));
    for(int i = 0; i < ranges.size(); i++) {
        umRanges[i] = std::make_pair(ranges[i].first, ranges[i].second);
    }

    // Allocate unified memory for text
    hipMallocManaged(&umText, text.length() * sizeof(char));
    for(int i = 0; i < text.length(); i++) {
        umText[i] = text[i];
    }

    // Allocate unified memory for pattern
    hipMallocManaged(&umPattern, pattern.length() * sizeof(char));
    for(int i = 0; i < pattern.length(); i++) {
        umPattern[i] = pattern[i];
    }
}

void RabinKarpSearch::calculateHashes()
{
    if(!hash)
        throw std::runtime_error("Hash not initialized.");

    std::tuple<long long, long long> hashes = hash
            ->forBase(BASE)
            .getPolyValues(pattern, text.substr(currentWindowPosition, pattern.length()));
    patternHash = std::get<0>(hashes) % PRIME;
    windowHash = std::get<1>(hashes) % PRIME;
}

void RabinKarpSearch::calculateRollingHash()
{
    windowHash = (windowHash - mostSignificantWeight * text[currentWindowPosition - 1]) * BASE + text[currentWindowPosition + pattern.length() - 1];
    windowHash = windowHash % PRIME;

    if(windowHash < 0)
    {
        windowHash = windowHash + PRIME;
    }
}

void RabinKarpSearch::moveWindow()
{
    ++currentWindowPosition;
}

std::vector<size_t> RabinKarpSearch::search()
{
    std::vector<size_t> matches;
    auto windows_count = text.length() - pattern.length();
    while(currentWindowPosition <= windows_count)
    {
        if(windowHash == patternHash)
        {
            if(pattern == text.substr(currentWindowPosition, pattern.length()))
            {
                //match found
                matches.push_back(currentWindowPosition);
            }
        }
        moveWindow();
        calculateRollingHash();
    }
    return matches;
}

__device__ bool RabinKarpSearch::compareCharArrays(const char *first, const char *second, int length) {
    int currentChar = 0;
    while(currentChar < length) {
        if(first[currentChar] != second[currentChar]) {
            return false;
        }
        return true;
    }
}

__global__ void searchForMatches(std::pair<int, int>* ranges, char* text, char* pattern) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    for(int i = ranges[idx].first; i <= ranges[idx].first; i++) {
        char* buff;

        free(buff);
    }
}

std::vector<size_t> RabinKarpSearch::searchGPU()
{
    searchForMatches<<<1, 32>>>(umRanges, umText, umPattern);
    hipDeviceSynchronize();
}