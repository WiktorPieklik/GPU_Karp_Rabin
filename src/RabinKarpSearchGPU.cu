#include "hip/hip_runtime.h"
#include "RabinKarpSearchGPU.cuh"

#include "Utility/TextSplitter.cuh"
#include "Text/TextFileReader.cuh"

#include "thrust/device_vector.h"

namespace {
    constexpr auto PRIME = 23;
    constexpr auto BASE = 36; //input alphabet's length
}

// Unified memory
std::pair<int, int>* umRanges;
int* umMatches;
char* umPattern;
char* umText;

// Variables
TextFileReader reader = TextFileReader("../test.txt");
std::string text = reader.read();
std::string pattern = "lorem";
int mostSignificantWeight = 1;
TextSplitter textSplitter = TextSplitter();
std::vector<std::pair<size_t, size_t>> ranges = textSplitter.splitText(text.length(), pattern.length());

// Allocate needed resources in unified memory
__host__ void initUnifiedMemory() {
    // Calculate most significant weight
    for(size_t i = 1; i < pattern.length(); ++i)
    {
        mostSignificantWeight *= BASE;
    }

    // Allocate unified memory for matches
    hipMallocManaged(&umMatches, (text.size() - pattern.length() + 1) * sizeof(int));
    for(int i = 0; i < (text.size() - pattern.length() + 1); i++) {
        umMatches[i] = 0;
    }

    // Allocate unified memory for ranges
    hipMallocManaged(&umRanges, ranges.size() * sizeof(std::pair<int, int>));
    for(int i = 0; i < ranges.size(); i++) {
        umRanges[i] = std::make_pair(ranges[i].first, ranges[i].second);
    }

    // Allocate unified memory for text
    hipMallocManaged(&umText, text.length() * sizeof(char));
    for(int i = 0; i < text.length(); i++) {
        umText[i] = text[i];
    }

    // Allocate unified memory for pattern
    hipMallocManaged(&umPattern, pattern.length() * sizeof(char));
    for(int i = 0; i < pattern.length(); i++) {
        umPattern[i] = pattern[i];
    }
}

__host__ __device__ long long getPolyValue(char* pattern, int pos, int len)
{
    long long result = static_cast<int>(pattern[pos]);
    for(size_t i = pos + 1; i < pos + len; ++i)
    {
        result = result * BASE + static_cast<int>(pattern[i]);
    }

    return result % PRIME;
}

__device__ long long calculateRollingHash(long long mostSignificantWeight, char* text, long long windowHash, int currentPos, int patternLen)
{
    long long hash = (windowHash - mostSignificantWeight * text[currentPos - 1]) * BASE + text[currentPos + patternLen - 1];
    hash = hash % PRIME;

    if(hash >= 0) {
        return hash;
    } else {
        return hash + PRIME;
    }
}

// Function that takes two char arrays and compares them
__device__ bool compareCharArrays(const char *first, const char *second, int pos, int length) {
    int currentChar = 0;
    while(currentChar < length) {
        if(first[currentChar] != second[pos + currentChar]) {
            return false;
        }
        currentChar++;
    }
    return true;
}

// GPU karp rabin search
__global__ void search(std::pair<int, int>* ranges, int* matches, char* text, char* pattern, int patternLen, long long patternHash, int mostSignificantWeight) {
    // Calculate unique index of a thread
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;

    // Variable for storing hash of the window
    long long windowHash;

    for(int i = ranges[threadId].first; i <= ranges[threadId].second; i++) {
        // Hash of first window
        if (i == ranges[threadId].first) {
            windowHash = getPolyValue(text, i, patternLen);
        }
        else {
            windowHash = calculateRollingHash(mostSignificantWeight, text, windowHash, i, patternLen);
        }

        if(windowHash == patternHash) {
            if (compareCharArrays(pattern, text, i, patternLen)) {
                matches[i]++;
            }
        }
    }
}

__host__ void printMatches() {
    for(int i = 0; i < (text.size() - pattern.length() + 1); i++) {
        // Print only indexes of matches
        if(umMatches[i] == 1) {
            std::cout<< i <<std::endl;
        }
    }
}

// Wrapper of kernel function to make it possible to use it with benchmark class
__host__ void searchWrapper() {
    // Calculate pattern hash value
    long long patternHash = getPolyValue(umPattern, 0, pattern.length());

    // Invoke kernel function
    search<<<1, 32>>>(umRanges, umMatches, umText, umPattern, pattern.length(), patternHash, mostSignificantWeight);

    // Make CPU wait for kernel to finish before go further
    hipDeviceSynchronize();
}

// Free the allocated resources
__host__ void freeUnifiedMemory() {
    hipFree(umRanges);
    hipFree(umText);
    hipFree(umPattern);
}
