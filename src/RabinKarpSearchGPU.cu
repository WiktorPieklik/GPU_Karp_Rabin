#include "hip/hip_runtime.h"
#include "RabinKarpSearchGPU.cuh"

#include "Utility/TextSplitter.cuh"
#include "Text/TextFileReader.cuh"

namespace {
    constexpr auto PRIME = 23;
    constexpr auto BASE = 36; //input alphabet's length
}

// Unified memory
std::pair<int, int>* umRanges;
char* umPattern;
char* umText;

// Variables
TextFileReader reader = TextFileReader("../test.txt");
std::string text = reader.read();
std::string pattern = "lorem";
int patternLen = pattern.length();
int mostSignificantWeight = 1;
TextSplitter textSplitter = TextSplitter();
std::vector<std::pair<size_t, size_t>> ranges = textSplitter.splitText(text.length(), pattern.length());

// Allocate needed resources in unified memory
__host__ void initUnifiedMemory() {
    // Calculate most significant weight
    for(size_t i = 1; i < pattern.length(); ++i)
    {
        mostSignificantWeight *= BASE;
    }

    // Allocate unified memory for ranges
    hipMallocManaged(&umRanges, ranges.size() * sizeof(std::pair<int, int>));
    for(int i = 0; i < ranges.size(); i++) {
        umRanges[i] = std::make_pair(ranges[i].first, ranges[i].second);
    }

    // Allocate unified memory for text
    hipMallocManaged(&umText, text.length() * sizeof(char));
    for(int i = 0; i < text.length(); i++) {
        umText[i] = text[i];
    }

    // Allocate unified memory for pattern
    hipMallocManaged(&umPattern, pattern.length() * sizeof(char));
    for(int i = 0; i < pattern.length(); i++) {
        umPattern[i] = pattern[i];
    }
}

__host__ __device__ long long getPolyValue(char* pattern, int pos, int len)
{
    long long result = static_cast<int>(pattern[pos]);
    for(size_t i = pos; i < pos + len; ++i)
    {
        result = result * BASE + static_cast<int>(pattern[i]);
    }

    return result % PRIME;
}

__device__ long long calculateRollingHash(long long mostSignificantWeight, char* text, long long windowHash, int currentPos, int patternLen)
{
    long long hash = (windowHash - mostSignificantWeight * text[currentPos - 1]) * BASE + text[currentPos + patternLen - 1];
    hash = hash % PRIME;
    if(hash >= 0) {
        return hash;
    } else {
        return hash + PRIME;
    }
}

// Function that takes two char arrays and compares them
__device__ bool compareCharArrays(const char *first, const char *second, int pos, int length) {
    int currentChar = 0;
    while(currentChar < length) {
        if(first[currentChar] != second[pos + currentChar]) {
            return false;
        }
        currentChar++;
    }
    return true;
}

// GPU karp rabin search
__global__ void search(std::pair<int, int>* ranges, char* text, char* pattern, int patternLen, long long patternHash, int mostSignificantWeight) {
    // Calculate unique index of a thread
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if hash was calculated at first
    bool isHashCalculated = false;

    long long windowHash;

    for(int i = ranges[idx].first; i <= ranges[idx].second; i++) {
        // Hash of first window
        if (isHashCalculated == false) {
            windowHash = getPolyValue(text, i, patternLen);
            isHashCalculated = true;
        }
        else {
            windowHash = calculateRollingHash(mostSignificantWeight, text, windowHash, i, patternLen);
        }
        for (int j = i; j < i + patternLen; j++) {
            printf("%c", text[j]);
        }
        printf("%lld\n", windowHash);
        if(windowHash == patternHash) {
            if (compareCharArrays(pattern, text, i, patternLen)) {
                printf("%d\n", i);
            }
        }
    }
}

// Wrapper of kernel function to make it possible to use it with benchmark class
__host__ void searchWrapper() {
    // Calculate pattern hash value
    long long patternHash = getPolyValue(umPattern, 0, patternLen);

    // Invoke kernel function
    search<<<1, 1>>>(umRanges, umText, umPattern, patternLen, patternHash, mostSignificantWeight);

    // Make CPU wait for kernel to finish before go further
    hipDeviceSynchronize();
}

// Free the allocated resources
__host__ void freeUnifiedMemory() {
    hipFree(umRanges);
    hipFree(umText);
    hipFree(umPattern);
}
