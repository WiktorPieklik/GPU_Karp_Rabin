#include "hip/hip_runtime.h"
#include <iostream>
#include "Hash/StandardHash.cuh"
#include "header/RabinKarpSearch.cuh"
#include "Text/FileNotFoundException.cuh"

// Function that takes two char arrays and compares them
__host__ __device__ bool compareCharArrays(const char *first, const char *second, int length) {
    int currentChar = 0;
    while(currentChar < length) {
        if(first[currentChar] != second[currentChar]) {
            return false;
        }
        currentChar++;
    }
    return true;

}

// GPU karp rabin search
__global__ void searchForMatches(std::pair<int, int>* ranges, char* text, char* pattern) {\
    // Calculate unique index of a thread
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    for(int i = ranges[idx].first; i <= ranges[idx].second; i++) {
        char* substr = (char*)malloc(6 * sizeof(char));
        memcpy(substr, &text[i], 5);
        substr[6] = '\0';
        if(compareCharArrays(pattern, substr, 6)) {
            printf("%s %d\n", substr, i);
        }
        free(substr);
    }
}

int main()
{
    // Unified memory
    std::pair<int, int>* umRanges;
    char* umPattern;
    char* umText;

    // Create TextSplitter object
    TextSplitter textSplitter = TextSplitter();
    auto textProcessor = RabinKarpSearch("../test.txt", "lorem", std::make_unique<StandardHash>());
    std::vector<size_t> matchess = textProcessor.search();
    for (auto i : matchess) {
        std::cout << i << std::endl;
    }
    std::vector<size_t> matches = textProcessor.search();
    std::string text = textProcessor.getText();
    std::string pattern = textProcessor.getPattern();

    // Allocate unified memory for ranges
    std::vector<std::pair<size_t, size_t>> ranges = textSplitter.splitText(text.length(), pattern.length());
    hipMallocManaged(&umRanges, ranges.size() * sizeof(std::pair<int, int>));
    for(int i = 0; i < ranges.size(); i++) {
        umRanges[i] = std::make_pair(ranges[i].first, ranges[i].second);
    }

    // Allocate unified memory for text
    umText = new char[text.length() * sizeof(char)];
    hipMallocManaged(&umText, text.length() * sizeof(char));
    for(int i = 0; i < text.length(); i++) {
        umText[i] = text[i];
    }

    // Allocate unified memory for pattern
    hipMallocManaged(&umPattern, pattern.length() * sizeof(char));
    for(int i = 0; i < pattern.length(); i++) {
        umPattern[i] = pattern[i];
    }

    // Set heap size (128mb). It's a random size, probably much to big :D
    hipDeviceSetLimit(hipLimitMallocHeapSize, 128*1024*1024);

    // Invoke kernel function
    searchForMatches<<<1, 32>>>(umRanges, umText, umPattern);

    // Make CPU wait for kernel to finish before go further
    hipDeviceSynchronize();

    // Free the allocated resources
    hipFree(umRanges);
    hipFree(umText);
    hipFree(umPattern);

    return 0;
}
